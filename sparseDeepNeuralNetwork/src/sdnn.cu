#include "hip/hip_runtime.h"
#include "common.cuh"
#include "graph.hpp"
#include "ref.cuh"
// #include "intrinsics.cuh"
#include <gflags/gflags.h>
#include <hipsparse.h>
#include <algorithm>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime_api.h>

#define BLOCK_SIZE 256

// template __global__ void csrmcsc_to_dense<1024>(int *csr_r, int *csr_c, double *csr_w, int *csc_c, int *csc_r, double *csc_w, double *out, int m, int n, int ldx);
// template __global__ void csrmcsc_to_dense<2048>(int *csr_r, int *csr_c, double *csr_w, int *csc_c, int *csc_r, double *csc_w, double *out, int m, int n, int ldx);
// template __global__ void csrmcsc_to_dense<4096>(int *csr_r, int *csr_c, double *csr_w, int *csc_c, int *csc_r, double *csc_w, double *out, int m, int n, int ldx);

// template void printD<float>(float *DeviceData, int n);

DEFINE_string(input, "", "dataset location");
DEFINE_int32(nn, 1024, "Neural number per layer");
DEFINE_int32(nl, 120, "layer number");

int main(int argc, char **argv)
{
    gflags::ParseCommandLineFlags(&argc, &argv, true);
    string inputFile;
    string categoryFile;
    string layerFile;
    if (FLAGS_input == "")
    {
        inputFile = "/home/pywang/graphchallenge/sparse-images-";
        categoryFile = "/home/pywang/graphchallenge/neuron";
        layerFile = "/home/pywang/graphchallenge/neuron";
    }
    else
    {
        inputFile = FLAGS_input + "/sparse-images-";
        categoryFile = FLAGS_input + "/neuron";
        layerFile = FLAGS_input + "/neuron";
    }
    int nNeuronPerLayer = FLAGS_nn;
    int nLayer = FLAGS_nl;
    double neuralNetBias[4] = {-0.3, -0.35, -0.4, -0.45};
    double bias;
    switch (nNeuronPerLayer)
    {
    case 1024:
    {
        bias = neuralNetBias[0];
        break;
    }
    case 4096:
    {
        bias = neuralNetBias[1];
        break;
    }
    case 16384:
    {
        bias = neuralNetBias[2];
        break;
    }
    case 65536:
    {
        bias = neuralNetBias[3];
        break;
    }
    }

    hipSetDevice(0);
    hipFree(0);

    csrgemm2Info_t info = NULL;
    hipsparseHandle_t handle = 0;
    hipsparseCreate(&handle);
    hipsparseCreateCsrgemm2Info(&info);
    hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);

    stringstream ss;
    ss << inputFile << nNeuronPerLayer << ".gr";
    string feature_name = ss.str();
    ss.str("");
    // ss.clear();
    cout << "read file start" << endl;
    Graph feature_vector(feature_name);
    feature_vector.Read();
    int nSample = 60000;
    int batch = nSample;

    SparseMat featureVectors;
    featureVectors.csrRowPtrA = feature_vector.xadj;
    featureVectors.csrColIndA = feature_vector.adjncy;
    featureVectors.csrValA = feature_vector.adjwgt;
    featureVectors.total_non_zero = feature_vector.num_edges;
    featureVectors.num_rows = feature_vector.num_nodes;

    SparseCSC *layers = new SparseCSC[nLayer];

    for (int i = 0; i < nLayer; i++)
    {
        ss << layerFile << nNeuronPerLayer << "/n" << nNeuronPerLayer << "-l" << (i + 1) << ".gr";
        string layer_name = ss.str();
        ss.str("");
        Graph layer_vector(layer_name);
        // layer_vector.ReadGraphCSC();
        layer_vector.ReadCSC();
        layers[i].cscColPtrA = layer_vector.csc_xadj;
        layers[i].cscRowIndA = layer_vector.csc_adjncy;
        layers[i].cscValA = layer_vector.csc_adjwgt;
        layers[i].total_non_zero = layer_vector.num_edges;
        layers[i].num_cols = layer_vector.num_nodes;
    }

    int nNeuron = nNeuronPerLayer;
    malloccopyCSR2Device(&featureVectors);
    SparseMat &spmA = featureVectors;
    // SparseMat spmA[feature_vector.s_n];
    SparseCSC spmB;
    // SparseMat spmC;
    // cout << "feature_vector finished" << endl;

    long long DNNedges = 0;
    int bnnz = layers[0].total_non_zero;
    for (size_t i = 0; i < nLayer; i++)
    {
        DNNedges += layers[i].total_non_zero;
        if (bnnz < layers[i].total_non_zero)
        {
            bnnz = layers[i].total_non_zero;
        }
    }
    // spmB.capacity = bnnz;
    spmB.total_non_zero = bnnz;
    spmB.num_cols = nNeuron;
    mallocCSCDevice(&spmB);
    // cout << "spmA data memory usage: " << (double)((spmA.num_rows + 1 + spmA.total_non_zero) * sizeof(int) + spmA.total_non_zero * sizeof(double)) / 1024 / 1024 << " MB" << endl;
    // cout << "tmp layer data memory usage: " << (double)((spmB.num_cols + 1 + spmB.total_non_zero) * sizeof(int) + spmB.total_non_zero * sizeof(double)) / 1024 / 1024 << " MB" << endl;

    SparseMat spmC;
    spmC.num_rows = batch;

    int tmp_nnz_h;
    int *tmp_nnz_per_row, *index;
    CudaSafeCall(hipMallocManaged(&(tmp_nnz_per_row), batch * sizeof(int)));
    CudaSafeCall(hipMalloc(&(index), (batch + 1) * sizeof(int)));

    // uint nstreams = 5;
    // hipStream_t *streams = (hipStream_t *)malloc(nstreams * sizeof(hipStream_t));
    // for (int i = 0; i < nstreams; i++)
    // {
    //     gpuErrorcheck(hipStreamCreate(&(streams[i])));
    // }

    hipProfilerStart();
    Timer p;
    p.Start();
    for (size_t i = 0; i < nLayer; i++)
    {
        void *d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        spmC.total_non_zero = spmA.total_non_zero;
        spmC.capacity = spmC.total_non_zero;
        mallocCSRDevice(&spmC);
        copyCSC2D(&layers[i], &spmB);
        CudaSafeCall(hipMemset(tmp_nnz_per_row, 0, batch * sizeof(int)));

        csrmcsc_nnz<<<spmA.num_rows, 1024>>>(spmA.csrRowPtrA_device, spmA.csrColIndA_device, spmA.csrValA_device,
                                             spmB.cscColPtrA_device, spmB.cscRowIndA_device, spmB.cscValA_device,
                                             tmp_nnz_per_row, spmA.num_rows, nNeuron, batch, bias);

        CudaSafeCall(hipMemset(spmC.csrRowPtrA_device, 0, (batch + 1) * sizeof(int)));
        hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes,
                                      tmp_nnz_per_row, spmC.csrRowPtrA_device + 1, spmC.num_rows);
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        hipcub::DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes,
                                      tmp_nnz_per_row, spmC.csrRowPtrA_device + 1, spmC.num_rows);

        CudaSafeCall(hipMemcpy(&spmC.total_non_zero, spmC.csrRowPtrA_device + spmC.num_rows,
                                sizeof(int), hipMemcpyDeviceToHost));
        if (spmC.total_non_zero > spmC.capacity)
            reallocate(&spmC, spmC.total_non_zero);
        CudaSafeCall(hipMemcpy(index, spmC.csrRowPtrA_device,
                                (batch + 1) * sizeof(int), hipMemcpyDeviceToDevice));

        csrmcsc_compute<<<spmA.num_rows * 32 / BLOCK_SIZE + 1, BLOCK_SIZE>>>(spmA.csrRowPtrA_device, spmA.csrColIndA_device, spmA.csrValA_device,
                                                                             spmB.cscColPtrA_device, spmB.cscRowIndA_device, spmB.cscValA_device,
                                                                             spmC.csrRowPtrA_device, spmC.csrColIndA_device, spmC.csrValA_device,
                                                                             index, spmA.num_rows, nNeuron, batch, bias);
        {
            // cout << "\nspmA.num_rows " << spmA.num_rows;
            // cout << "\nspmA " << endl;
            // printD<int>(spmC.csrRowPtrA_device, 9);
            // cout << endl;
            printD<int>(spmC.csrColIndA_device,spmC.total_non_zero);
            cout << endl;
            printD<double>(spmC.csrValA_device, spmC.total_non_zero);
            return 0;
        }
        repoint(&spmA, &spmC);
        CudaSafeCall(hipFree(d_temp_storage));
        // if (i % 50 == 0)
        //     cout << "layer " << i << " compute finished with nnz\t" << spmA.total_non_zero << endl;
    }
    hipProfilerStop();
    double t = p.Finish() / 1000;
    cout << "DNN neurons/layer: " << nNeuron << " , layers: " << nLayer << endl;
    cout << "Run time (sec): " << t << ", run rate (edges/sec): " << nSample * DNNedges / t << endl;

    vector<int> out_nnz;
    int nnz = 0;

    copyDeviceCSR2Host(&spmA);
    nnz += spmA.total_non_zero;

    Timer tr;
    tr.Start();

    for (size_t j = 0; j < spmA.num_rows; j++)
    {
        double tmp = 0;
        for (size_t k = spmA.csrRowPtrA[j]; k < spmA.csrRowPtrA[j + 1]; k++)
        {
            if (spmA.csrValA[k] > 0)
            {
                out_nnz.push_back(j);
                break;
            }
        }
    }
    // cout << "\nNumber of categories  " << out_nnz.size() << endl;
    // cout << " Identify the categories finished using " << tr.Finish() / 1000 << " s\n";
    // cout << "computed categories of first 10 \n";
    // for (size_t i = 0; i < 10; i++)
    // {
    //     cout << out_nnz[i] + 1 << "\t";
    // }
    SparseMat result;
    ss.str("");
    ss << categoryFile << nNeuronPerLayer << "-l" << nLayer << "-categories.tsv";
    string true_files = ss.str();
    ifstream tf;
    tf.open(true_files);

    vector<int> ground_truth;
    string line;
    int val = 0;
    while (getline(tf, line))
    {
        ss.str("");
        ss.clear();
        ss << line;
        ss >> val;
        ground_truth.push_back(val);
    }

    // cout << "\ntrue categories of first 10 \n";
    // for (size_t i = 0; i < 10; i++)
    // {
    //     cout << ground_truth[i] << "\t";
    // }
    bool pass = true;
    for (size_t i = 0; i < ground_truth.size(); i++)
    {
        if (ground_truth[i] != out_nnz[i] + 1)
        {
            pass = false;
            // cout << "At " << i << "ground_truth[i]:\t" << ground_truth[i] << "out_nnz[i]\t" << out_nnz[i] + 1 << endl;
        }
    }
    if (pass)
    {
        cout << "Challenge PASSED ";
    }
    else
        cout << "Challenge failed ";

    return 0;
}
